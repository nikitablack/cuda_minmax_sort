#include <opencv2/cudaarithm.hpp>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <stdexcept>
#include <string>

#define CHECK_ERROR(ans)                       \
    {                                          \
        checkError((ans), __FILE__, __LINE__); \
    }

inline void checkError(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        printf("GPU Kernel Error: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort)
        {
            throw std::runtime_error{"Cuda error " + std::to_string(code)};
        }
    }
}

int main()
{
    cv::cuda::GpuMat m(1000, 1000, CV_32FC1);

    for (uint32_t i{0}; i < 10; ++i)
    {
        double maxValue;
        cv::cuda::minMax(m, nullptr, &maxValue);
    }

    constexpr size_t n{1'000'000};

    int *id;
    float *a;
    float *b;
    float *c;
    CHECK_ERROR(hipMalloc(&id, n * sizeof(int)));
    CHECK_ERROR(hipMalloc(&a, n * sizeof(float)));
    CHECK_ERROR(hipMalloc(&b, n * sizeof(float)));
    CHECK_ERROR(hipMalloc(&c, n * sizeof(float)));

    thrust::device_ptr<int> id_ptr(id);
    thrust::device_ptr<float> a_ptr(a);
    thrust::device_ptr<float> b_ptr(b);
    thrust::device_ptr<float> c_ptr(c);

    auto it = thrust::make_zip_iterator(thrust::make_tuple(a_ptr, b_ptr, c_ptr));

    for (uint32_t i{0}; i < 10; ++i)
    {
        thrust::sort_by_key(id_ptr, id_ptr + n, it);
    }

    CHECK_ERROR(hipFree(id));
    CHECK_ERROR(hipFree(a));
    CHECK_ERROR(hipFree(b));
    CHECK_ERROR(hipFree(c));

    return 0;
}